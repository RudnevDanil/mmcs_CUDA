#include "hip/hip_runtime.h"
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <iostream>
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

using namespace cv;
using namespace std;

#define CHECK(value) {                                          \
    hipError_t _m_cudaStat = value;                                        \
    if (_m_cudaStat != hipSuccess) {                                       \
        cout<< "Error:" << hipGetErrorString(_m_cudaStat) \
            << " at line " << __LINE__ << " in file " << __FILE__ << "\n"; \
        exit(1);                                                            \
    } }




__global__ void cudoCalculculate(unsigned char* image,int h,int w){
   
    int i = 3 * (threadIdx.x + blockIdx.x * blockDim.x);
    // �������� ������� �����
    image[i+2] = 0;
       
}



void CPU() {

    Mat image;
    image = cv::imread("pic.jpg", cv::IMREAD_COLOR);   // Read the file CV_LOAD_IMAGE_COLOR
    if (!image.data)                              // Check for invalid input
    {
        cout << "Could not open or find the image" << std::endl;
        return;
    }
    Mat result = image.clone();
    clock_t start = clock();
    for (int i = 0; i < image.rows; i++)
    {
        //pointer to 1st pixel in row
        Vec3b* p = image.ptr<Vec3b>(i);
        Vec3b* p_r = result.ptr<Vec3b>(i);
        for (int j = 0; j < image.cols; j++)
            //for (int ch = 0; ch < 3; ch++)
            p_r[j][2] = 0;
    }
    clock_t end = clock();
    double seconds = (double)(end - start) / CLOCKS_PER_SEC;
    cout << "CPU time: " << seconds * 1000 << "ms" << endl;
    imwrite("pic2.jpg", result);
    //show image
    namedWindow("Display window", WINDOW_AUTOSIZE);// Create a window for display.
    imshow("Display window", result);                   // Show our image inside it.
    cv::waitKey(0);// Wait for a keystroke in the window  
}

void GPU() {
    Mat image;
    image = cv::imread("pic.jpg", cv::IMREAD_COLOR);   // Read the file CV_LOAD_IMAGE_COLOR
    if (!image.data)                              // Check for invalid input
    {
        cout << "Could not open or find the image" << std::endl;
        return;
    }
    unsigned char* imageGray;
    int full_size_image = image.rows * image.cols * 3;
    int h = image.cols;
    int w = image.rows;
    hipEvent_t startCUDA, stopCUDA;
    float elapsedTimeCUDA;
    hipEventCreate(&startCUDA);
    hipEventCreate(&stopCUDA);

    CHECK(hipMalloc(&imageGray, full_size_image));

    CHECK(hipMemcpy(imageGray, image.data, full_size_image, hipMemcpyHostToDevice));

    hipEventRecord(startCUDA, 0);

    cudoCalculculate <<<(full_size_image / 3 + 255) / 256, 256 >>> (imageGray, h,w);

    hipEventRecord(stopCUDA, 0);
    hipEventSynchronize(stopCUDA);
    CHECK(hipGetLastError());

    hipEventElapsedTime(&elapsedTimeCUDA, startCUDA, stopCUDA);

    cout << "CUDA sum time = " << elapsedTimeCUDA << " ms\n";
    cout << "CUDA memory throughput = " << 3 * full_size_image * sizeof(float) / elapsedTimeCUDA / 1024 / 1024 / 1.024 << " Gb/s\n";

    CHECK(hipMemcpy(image.data, imageGray, full_size_image, hipMemcpyDeviceToHost));
    CHECK(hipFree(imageGray));
    imwrite("pic2GPU.jpg", image);
    namedWindow("Display window", WINDOW_AUTOSIZE);// Create a window for display.
    imshow("Display window", image);                   // Show our image inside it.
    waitKey(0);

}

int main( int argc, char** argv )
{
    GPU();
    CPU();
    return 0;
}
